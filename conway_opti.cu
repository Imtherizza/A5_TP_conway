#include "hip/hip_runtime.h"
#include "conway.h"
#include <c10/cuda/CUDAStream.h>
#include <torch/extension.h>

__global__ void game_of_life_kernel(int *grid, int *new_grid, int width,
                                    int height) {
  for (int block_start_x = blockIdx.x * blockDim.x; block_start_x < width;
       block_start_x += blockDim.x * gridDim.x) {

    for (int block_start_y = blockIdx.y * blockDim.y; block_start_y < height;
         block_start_y += blockDim.y * gridDim.y) {

      int x = block_start_x + threadIdx.x;
      int y = block_start_y + threadIdx.y;


      // TODO: 1. Calculate the number of alive neighbors
      int alive = 0;
      for(int j = -1; j < 2; j++){
        for(int i = -1; i < 2; i++){
          if(!((x+i) < 0 || x+i> width-1 || y+j<0 || y+j>height-1 || (!i && !j)))alive += grid[(x+i)+(y*width+j*width)];
        }
      }

      // TODO: 2. Apply the rules of Conway's Game of Life
      new_grid[x+y*width] =((!(grid[x+y*width]) && (alive == 3)) || ((grid[x+y*width]) && !(alive < 2 || alive > 3)));


      // TODO: 3. Write the result to the new grid
      //new_grid = grid;

      // TODO(once you pass the conformance test): measure with nvprof, and
      // check for different ways of improving performance
    }
  }
}

void game_of_life_step(torch::Tensor grid_in, torch::Tensor grid_out,
                       std::optional<torch::Stream> stream) {
  int width = grid_in.size(1);
  int height = grid_in.size(0);
  assert(grid_in.sizes() == grid_out.sizes());

  hipStream_t cudaStream = 0;
  if (stream.has_value()) {
    cudaStream = c10::cuda::CUDAStream(stream.value()).stream();
  }

  #define WARP_SIZE 32

  const dim3 blockSize(WARP_SIZE, WARP_SIZE);
  const dim3 gridSize(width/WARP_SIZE,height/WARP_SIZE);

  game_of_life_kernel<<<gridSize, blockSize, 0, cudaStream>>>(
      grid_in.data_ptr<int>(), grid_out.data_ptr<int>(), width, height);
}
