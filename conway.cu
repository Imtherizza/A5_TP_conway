#include "hip/hip_runtime.h"
#include "conway.h"
#include <c10/cuda/CUDAStream.h>
#include <torch/extension.h>

__global__ void game_of_life_kernel(int *grid, int *new_grid, int width,
                                    int height) {
  for (int block_start_x = blockIdx.x * blockDim.x; block_start_x < width;
       block_start_x += blockDim.x * gridDim.x) {

    for (int block_start_y = blockIdx.y * blockDim.y; block_start_y < height;
         block_start_y += blockDim.y * gridDim.y) {

      int x = block_start_x + threadIdx.x;
      int y = block_start_y + threadIdx.y;

      if (x >= width || y >= height)
        continue;

      // TODO: 1. Calculate the number of alive neighbors
      // TODO: 2. Apply the rules of Conway's Game of Life
      // TODO: 3. Write the result to the new grid

      // TODO(once you pass the conformance test): measure with nvprof, and
      // check for different ways of improving performance
    }
  }
}

void game_of_life_step(torch::Tensor grid_in, torch::Tensor grid_out,
                       std::optional<torch::Stream> stream) {
  int width = grid_in.size(1);
  int height = grid_in.size(0);
  assert(grid_in.sizes() == grid_out.sizes());

  hipStream_t cudaStream = 0;
  if (stream.has_value()) {
    cudaStream = c10::cuda::CUDAStream(stream.value()).stream();
  }

  const dim3 blockSize(1, 16);
  const dim3 gridSize(1, 1);

  game_of_life_kernel<<<gridSize, blockSize, 0, cudaStream>>>(
      grid_in.data_ptr<int>(), grid_out.data_ptr<int>(), width, height);
}
