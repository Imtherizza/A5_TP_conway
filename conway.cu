#include "hip/hip_runtime.h"
#include "conway.h"
#include <c10/cuda/CUDAStream.h>
#include <torch/extension.h>

__global__ void game_of_life_kernel(signed char *grid, signed char *new_grid, short width,
                                    short height) {
  for (int block_start_y = blockIdx.y * blockDim.y; block_start_y < height;
         block_start_y += blockDim.y * gridDim.y) {
  
    for (int block_start_x = blockIdx.x * blockDim.x; block_start_x < width;
        block_start_x += blockDim.x * gridDim.x) {

      short x = block_start_x + threadIdx.x;
      short y = block_start_y + threadIdx.y;


      // TODO: 1. Calculate the number of alive neighbors
      char alive = 0;
      for(char j = -1; j < 2; j++){
        for(char i = -1; i < 2; i++){
          if(!((x+i) < 0 || x+i> width-1 || y+j<0 || y+j>height-1 || (!i && !j)))alive += grid[(x+i)+(y*width+j*width)];
        }
      }

      // TODO: 2. Apply the rules of Conway's Game of Life
      new_grid[x+y*width] =((!(grid[x+y*width]) && (alive == 3)) || ((grid[x+y*width]) && !(alive < 2 || alive > 3)));

    }
  }
}

void game_of_life_step(torch::Tensor grid_in, torch::Tensor grid_out,
                       std::optional<torch::Stream> stream) {
  char width = grid_in.size(1);
  char height = grid_in.size(0);
  assert(grid_in.sizes() == grid_out.sizes());

  hipStream_t cudaStream = 0;
  if (stream.has_value()) {
    cudaStream = c10::cuda::CUDAStream(stream.value()).stream();
  }

  #define WARP_SIZE 32

  // const dim3 blockSize( WARP_SIZE,WARP_SIZE);
  const dim3 blockSize( WARP_SIZE,3);
  const dim3 gridSize(width/WARP_SIZE + 1,height/3 + 1);

  game_of_life_kernel<<<gridSize, blockSize, 0, cudaStream>>>(
      grid_in.data_ptr<signed char>(), grid_out.data_ptr<signed char>(), width, height);
}
