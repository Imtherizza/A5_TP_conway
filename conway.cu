#include "hip/hip_runtime.h"
#include "conway.h"
#include <c10/cuda/CUDAStream.h>
#include <torch/extension.h>

__global__ void game_of_life_kernel(int *grid, int *new_grid, int width,
                                    int height) {
  for (int block_start_x = blockIdx.x * blockDim.x; block_start_x < width;
       block_start_x += blockDim.x * gridDim.x) {

    for (int block_start_y = blockIdx.y * blockDim.y; block_start_y < height;
         block_start_y += blockDim.y * gridDim.y) {

      int x = block_start_x + threadIdx.x;
      int y = block_start_y + threadIdx.y;

      if (x >= width || y >= height);
        //continue;

      // TODO: 1. Calculate the number of alive neighbors
      int alive = 0;
      for(int i = -1; i < 2; i++){
        for(int j = -1; j < 2; j++){

          if((x+i) < 0 || x+i> width-1 || y+j<0 || y+j>height-1 || (i == 0 && j == 0)){}
          else{
            alive += grid[(x+i)+(y*width+j*width)];
          }
        }
      }

      // TODO: 2. Apply the rules of Conway's Game of Life
      if(grid[x+y*width]){ // LIVE
        if (alive < 2 || alive > 3){ // KILL
          new_grid[x+y*width] = 0;
        } else {
          new_grid[x+y*width] = 1;
        }
      } else { // DED
        if(alive == 3){ // BORN
          new_grid[x+y*width] = 1;
        } else {
          new_grid[x+y*width] = 0;
        }
      }

      // TODO: 3. Write the result to the new grid
      //new_grid = grid;

      // TODO(once you pass the conformance test): measure with nvprof, and
      // check for different ways of improving performance
    }
  }
}

void game_of_life_step(torch::Tensor grid_in, torch::Tensor grid_out,
                       std::optional<torch::Stream> stream) {
  int width = grid_in.size(1);
  int height = grid_in.size(0);
  assert(grid_in.sizes() == grid_out.sizes());

  hipStream_t cudaStream = 0;
  if (stream.has_value()) {
    cudaStream = c10::cuda::CUDAStream(stream.value()).stream();
  }

  const dim3 blockSize(1, 16);
  const dim3 gridSize(1, 1);

  game_of_life_kernel<<<gridSize, blockSize, 0, cudaStream>>>(
      grid_in.data_ptr<int>(), grid_out.data_ptr<int>(), width, height);
}
